
#include <hip/hip_runtime.h>
#include <cstdio>

/*
	diameter: 2*Radius + 1

	0 0 0 0 0 0 0 0
	x x x x x x x x
	x x x x x x x x
	x x x x x x x x
	x x x x x x x x
	x x x x x x x x
	x x x x x x x x
	0 0 0 0 0 0 0 0 

	x - data
	0 - redundant halo
*/
template<int radius, int diameter>
__global__ void dFdY_kernel(const float * input, float * output, int nx, int ny)
{
	/// sm_20: 13 registers used by this kernel

	const int gtidx = blockDim.x * blockIdx.x + threadIdx.x;

	int outputIndex = gtidx + radius * nx;
	int inputIndex  = outputIndex - radius * nx;

	// declare local register buffer
	float buffer[diameter];

	// Fill the buffer up to start computations
	#pragma unroll
	for (int i = 1; i < diameter; ++i)
	{
		buffer[i] = input[inputIndex];
		inputIndex += nx;
	}

	/// Move front towards y (vertical) direction
	for (int y = 0; y < ny; ++y)
	{
		// update register values
		#pragma unroll
		for (int i = 0; i < diameter - 1; ++i)
		{
			buffer[i] = buffer[i + 1];
		}
		buffer[diameter - 1] = input[inputIndex];

		// compute (df/dy)(x,y) := [f(x,y+h) - f(x,y+h)]/2*h
		float derivative = 0.5f * (buffer[2] - buffer[0]);

		// write output
		output[outputIndex] = derivative;

		outputIndex += nx;
		inputIndex += nx;
	}
}


void TestPartialDerivative_dFdY()
{
	const int nx       = 2048;
	const int ny       = 2048;

	const int paddedny = (1 + ny + 1);

	const int nelem  = nx * paddedny;
	const int nbytes = nelem * sizeof(float);


	float* fh; hipHostMalloc((void**)&fh, nbytes, hipHostMallocDefault);
	float* dh; hipHostMalloc((void**)&dh, nbytes, hipHostMallocDefault);


	/// Fill input array: f(x,y) := (x-A)^2 + (y-B)^2

	/// Fill first halo row
	int a = -1;
	for (int x = 0; x < nx; ++x)
	{
		fh[x + (a + 1) * nx] = (float)((x - nx * 0.5f)*(x - nx * 0.5f) + (a - ny * 0.5f)*(a - ny * 0.5f));
	}
	/// Fill working data (function values)
	for (int y = 0; y < ny; ++y)
	{
		for (int x = 0; x < nx; ++x)
		{
			fh[x + (y + 1) * nx] = (float)((x - nx * 0.5f)*(x - nx * 0.5f) + (y - ny * 0.5f)*(y - ny * 0.5f));
		}
	}
	/// Fill last halo row
	a = ny;
	for (int x = 0; x < nx; ++x)
	{
		fh[x + (a + 1) * nx] = (float)((x - nx * 0.5f)*(x - nx * 0.5f) + (a - ny * 0.5f)*(a - ny * 0.5f));
	}

	/// Fill output array with zeros
	for (int y = 0; y < paddedny; ++y)
	{
		for (int x = 0; x < nx; ++x)
		{
			dh[x + y * nx] = 0.0f;
		}
	}

	float* fd; hipMalloc((void**)&fd, nbytes);
	float* dd; hipMalloc((void**)&dd, nbytes);

	hipMemcpy(fd, fh, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(dd, dh, nbytes, hipMemcpyHostToDevice);

	/// Initialize timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int nthread = 128;
	int nblock  = nx/nthread;

	/// Record time befor kernel launch
	hipEventRecord(start, 0);

	const int radius = 1;
	dFdY_kernel<radius, 2*radius + 1><<<nblock, nthread, (nthread + 2 * radius) * sizeof(float)>>>(fd, dd, nx, ny);

	/// Record time after simulation
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipMemcpy(dh, dd, nbytes, hipMemcpyDeviceToHost);


	/// Calculate kernel time
	float time_ms; hipEventElapsedTime(&time_ms, start, stop);
	printf("df/dy kernel time: %f ms\n", time_ms);

	/// Release timer
	hipEventDestroy(start);
	hipEventDestroy(stop);

	/// Free device memory
	hipFree(fd);
	hipFree(dd);

	/// write result to file
	FILE* file = fopen("resultDfDy.txt","w");

	for (int y = 0; y < paddedny; ++y)
	{
		for (int x = 0; x < nx; ++x)
		{
			fprintf(file,"%d %d %f \n", x, y, dh[x + y * nx]);
		}
		fprintf(file,"\n");
	}

	fclose(file);

	/// Free host memory
	hipHostFree(fh);
	hipHostFree(dh);
}


/*
    offset: 32 elements = 128B

	0 x x x x x x x x 0 
	0 x x x x x x x x 0 
	0 x x x x x x x x 0 
	0 x x x x x x x x 0 
	0 x x x x x x x x 0
	0 x x x x x x x x 0
	0 x x x x x x x x 0
	0 x x x x x x x x 0
	0 x x x x x x x x 0 

	x - data
	0 - redundant halo

	where 0 symbol means 128B offset (32*4B)
*/


template<int radius, int offset>
__global__ void dFdX_kernel(const float * input, float * output, int nx, int ny)
{
	/// sm_20: 14 registers used by this kernel

	extern __shared__ float smem[];

	const int gtidx     = blockDim.x * blockIdx.x + threadIdx.x;
	const int ltidx     = threadIdx.x;
	const int blockdimx = blockDim.x;
	const int rowsize   = offset + nx + offset;
	const int tx        = ltidx + radius;

	/// Move front towards y (vertical) direction
	for (int y = 0; y < ny; ++y)
	{
		// calculate global input index
		const int inputIndex = gtidx + offset + y * rowsize;

		__syncthreads();

		// load "halo" left && right
		if (ltidx < radius)
		{
			smem[ltidx] = input[inputIndex - radius];
			smem[ltidx + blockdimx + radius] = input[blockdimx + inputIndex];
		}

		// load "internal" data
		smem[tx] = input[inputIndex];

		__syncthreads();
		
		// compute (df/dx)(x,y) := [f(x+h,y) - f(x-h,y)]/2*h
		float derivative = 0.5f * (smem[tx + 1] - smem[tx - 1]);

		// write output
		output[inputIndex] = derivative;
	}
}


void TestPartialDerivative_dFdX()
{
	const int nx       = 2048;
	const int ny       = 2048;

	const int pad32    = 32;
	const int paddednx = (pad32 + nx + pad32);

	const int nelem  = paddednx * ny;
	const int nbytes = nelem * sizeof(float);


	float* fh; hipHostMalloc((void**)&fh, nbytes, hipHostMallocDefault);
	float* dh; hipHostMalloc((void**)&dh, nbytes, hipHostMallocDefault);
	memset(fh, 0, nbytes);
	memset(dh, 0, nbytes);

	/// Fill input array: f(x,y) := (x-A)^2 + (y-B)^2
	for (int y = 0; y < ny; ++y)
	{
		/// Fill first 32 elements in the row
		for (int x = 0; x < pad32; ++x)
		{
			fh[x + y * paddednx] = 0.0f;
		}
		int a = pad32 - 1;
		fh[a + y * paddednx] = (float)((a - paddednx * 0.5f)*(a - paddednx * 0.5f) + (y - ny * 0.5f)*(y - ny * 0.5f));
		/// Fill working data (function values)
		for (int x = pad32; x < pad32 + nx; ++x)
		{
			fh[x + y * paddednx] = (float)((x - paddednx * 0.5f)*(x - paddednx * 0.5f) + (y - ny * 0.5f)*(y - ny * 0.5f));
		}
		/// Fill last 32 elements in the row
		for (int x = pad32 + nx; x < pad32 + nx + pad32; ++x)
		{
			fh[x + y * paddednx] = 0.0f;
		}
		a = pad32 + nx;
		fh[a + y * paddednx] = (float)((a - paddednx * 0.5f)*(a - paddednx * 0.5f) + (y - ny * 0.5f)*(y - ny * 0.5f));
	}

	/// Fill output array with zeros
	for (int y = 0; y < ny; ++y)
	{
		for (int x = 0; x < paddednx; ++x)
		{
			dh[x + y * paddednx] = 0.0f;
		}
	}

	float* fd; hipMalloc((void**)&fd, nbytes);
	float* dd; hipMalloc((void**)&dd, nbytes);

	hipMemcpy(fd, fh, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(dd, dh, nbytes, hipMemcpyHostToDevice);

	/// Initialize timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	int nthread = 128;
	int nblock  = nx/nthread;

	/// Record time befor kernel launch
	hipEventRecord(start, 0);

	const int radius = 1;
	dFdX_kernel<radius, pad32><<<nblock, nthread, (nthread + 2*radius) * sizeof(float)>>>(fd, dd, nx, ny);

	/// Record time after simulation
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipMemcpy(dh, dd, nbytes, hipMemcpyDeviceToHost);

	/// Calculate kernel time
	float time_ms; hipEventElapsedTime(&time_ms, start, stop);
	printf("df/dx kernel time: %f ms\n", time_ms);

	/// Release timer
	hipEventDestroy(start);
	hipEventDestroy(stop);

	/// Free device memory
	hipFree(fd);
	hipFree(dd);

	/// write result to file
	FILE* file = fopen("resultDfDx.txt","w");

	for (int y = 0; y < ny; ++y)
	{
		for (int x = pad32; x < pad32 + nx; ++x)
		{
			fprintf(file,"%d %d %f \n", x, y, dh[x + y * paddednx]);
		}
		fprintf(file,"\n");
	}

	fclose(file);

	/// Free host memory
	hipHostFree(fh);
	hipHostFree(dh);
}


int main(int argc, char** argv)
{
	TestPartialDerivative_dFdY();
	TestPartialDerivative_dFdX();

	return 0;
}